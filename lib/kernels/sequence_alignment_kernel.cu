#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 Quim Aguado
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <stdbool.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "sequence_alignment_kernel.cuh"

#define MAX_PB(A, B) llmax((A), (B))
#define MAX(A, B) max((A), (B))
#define MIN(A, B) min((A), (B))

// At least one of the highest two bits is set
#define BT_WORD_FULL_CMP 0x40000000
#define BT_IS_FULL(bt_word) ((bt_word) >= BT_WORD_FULL_CMP)

__device__ wfa_offset_t WF_extend_kernel (const char* text,
                                  const char* pattern,
                                  const int tlen,
                                  const int plen,
                                  const int k,
                                  const wfa_offset_t offset_k) {
    int v  = EWAVEFRONT_V(k, offset_k);
    int h  = EWAVEFRONT_H(k, offset_k);

    const int bases_to_cmp = 16;
    int eq_elements = 0;
    int acc = 0;
    // Compare 16 bases at once
    while (v < plen && h < tlen) {
        // Which byte to pick
        int real_v = v / 4;
        int real_h = h / 4;

        // Get the displacement inside the aligned word
        int pattern_displacement = v % bases_to_cmp;
        int text_displacement = h % bases_to_cmp;

        // 0xffffffffffffff00
        uintptr_t alignment_mask = (uintptr_t)-1 << 2;
        uint32_t* word_p_ptr = (uint32_t*)((uintptr_t)(pattern + real_v) & alignment_mask);
        uint32_t* next_word_p_ptr = word_p_ptr + 1;
        uint32_t* word_t_ptr = (uint32_t*)((uintptr_t)(text + real_h) & alignment_mask);
        uint32_t* next_word_t_ptr = word_t_ptr + 1;


        // * 2 because each element is 2 bits
        uint32_t sub_word_p_1 = *word_p_ptr;
        uint32_t sub_word_p_2 = *next_word_p_ptr;
        sub_word_p_1 = sub_word_p_1 << (pattern_displacement * 2);
        // Convert the u32 to big-endian, as little endian inverts the order
        // for the sequences.
        sub_word_p_2 = *next_word_p_ptr;
        // Cast to uint64_t is done to avoid undefined behaviour in case
        // it's shifted by 32 elements.
        // ----
        // The type of the result is that of the promoted left operand. The
        // behavior is undefined if the right operand is negative, or
        // greater than or equal to the length in bits of the promoted left
        // operand.
        // ----
        sub_word_p_2 = ((uint64_t)sub_word_p_2) >>
            ((bases_to_cmp - pattern_displacement) * 2);

        uint32_t sub_word_t_1 = *word_t_ptr;
        sub_word_t_1 = sub_word_t_1 << (text_displacement * 2);
        uint32_t sub_word_t_2 = *next_word_t_ptr;
        sub_word_t_2 = ((uint64_t)sub_word_t_2) >>
            ((bases_to_cmp - text_displacement) * 2);

        uint32_t word_p = sub_word_p_1 | sub_word_p_2;
        uint32_t word_t = sub_word_t_1 | sub_word_t_2;

        uint32_t diff = word_p ^ word_t;
        // Branchless method to remove the equal bits if we read "too far away"
        uint32_t full_mask = (uint32_t)-1;
        int next_v = v + bases_to_cmp;
        int next_h = h + bases_to_cmp;
        uint32_t mask_p = full_mask << ((next_v - plen) * 2 * (next_v > plen));
        uint32_t mask_t = full_mask << ((next_h - tlen) * 2 * (next_h > tlen));
        diff = diff | ~mask_p | ~mask_t;

        int lz = __clz(diff);

        // each element has 2 bits
        eq_elements = lz / 2;
        acc += eq_elements;

        if (eq_elements < bases_to_cmp) {
            break;
        }


        v += bases_to_cmp;
        h += bases_to_cmp;
    }

    return offset_k + acc;
}

__device__ uint32_t offload_backtrace (unsigned int* const last_free_bt_position,
                                   const wfa_backtrace_t backtrace,
                                   wfa_backtrace_t* const global_backtraces_array) {
    uint32_t old_val = atomicAdd(last_free_bt_position, 1);

    global_backtraces_array[old_val].backtrace = backtrace.backtrace;
    global_backtraces_array[old_val].prev = backtrace.prev;

    //printf("(tid = %d) Offloading backtrace! old: %d\n", threadIdx.x, old_val);

    // TODO: Check if new_val is more than 32 bits
    return old_val;
}

__device__ void next_M (wfa_wavefront_t* M_wavefronts,
                        const int curr_wf,
                        const int active_working_set_size,
                        const int x,
                        const char* text,
                        const char* pattern,
                        const int tlen,
                        const int plen,
                        unsigned int* const last_free_bt_position,
                        wfa_backtrace_t* const offloaded_backtraces) {
    // The wavefront do not grow in case of mismatch
    const wfa_wavefront_t* prev_wf = &M_wavefronts[(curr_wf + x) % active_working_set_size];
    const int hi = prev_wf->hi;
    const int lo = prev_wf->lo;

    for (int k=lo + threadIdx.x; k <= hi; k+=blockDim.x) {
        wfa_offset_t curr_offset = prev_wf->offsets[k] + 1;

        curr_offset = WF_extend_kernel(text, pattern,
                                       tlen, plen, k, curr_offset);

        M_wavefronts[curr_wf].offsets[k] = curr_offset;

        wfa_backtrace_t prev_bt = prev_wf->backtraces[k];
        uint32_t backtrace_val = (prev_bt.backtrace << 2) | OP_SUB;
        uint32_t prev = prev_bt.prev;
        wfa_backtrace_t M_backtrace = {
            .backtrace = backtrace_val,
            .prev = prev
            };

        if (BT_IS_FULL(backtrace_val)) {
            prev = offload_backtrace(last_free_bt_position,
                                     M_backtrace,
                                     offloaded_backtraces);
            M_backtrace = {.backtrace = 0, .prev = prev};
        }

        M_wavefronts[curr_wf].backtraces[k] = M_backtrace;
    }

    if (threadIdx.x == 0) {
        M_wavefronts[curr_wf].hi = hi;
        M_wavefronts[curr_wf].lo = lo;
        M_wavefronts[curr_wf].exist= true;
    }
}

__device__ void next_MDI (wfa_wavefront_t* M_wavefronts,
                          wfa_wavefront_t* I_wavefronts,
                          wfa_wavefront_t* D_wavefronts,
                          const int curr_wf,
                          const int active_working_set_size,
                          const int x,
                          const int o,
                          const int e,
                          const char* text,
                          const char* pattern,
                          const int tlen,
                          const int plen,
                          unsigned int* const last_free_bt_position,
                          wfa_backtrace_t* const offloaded_backtraces) {
    const wfa_wavefront_t* prev_wf_x =   &M_wavefronts[(curr_wf + x) % active_working_set_size];
    const wfa_wavefront_t* prev_wf_o =   &M_wavefronts[(curr_wf + o + e) % active_working_set_size];
    const wfa_wavefront_t* prev_I_wf_e = &I_wavefronts[(curr_wf + e) % active_working_set_size];
    const wfa_wavefront_t* prev_D_wf_e = &D_wavefronts[(curr_wf + e) % active_working_set_size];

    const int hi_ID = MAX(prev_wf_o->hi, MAX(prev_I_wf_e->hi, prev_D_wf_e->hi)) + 1;
    const int hi    = MAX(prev_wf_x->hi, hi_ID);
    const int lo_ID = MIN(prev_wf_o->lo, MIN(prev_I_wf_e->lo, prev_D_wf_e->lo)) - 1;
    const int lo    = MIN(prev_wf_x->lo, lo_ID);

    for (int k=lo + threadIdx.x; k <= hi; k+=blockDim.x) {
        // ~I offsets
        const wfa_offset_t I_gap_open_offset = prev_wf_o->offsets[k - 1] + 1;
        const wfa_backtrace_t I_gap_open_bt = prev_wf_o->backtraces[k - 1];
        const int64_t I_gap_open_offset_pb = (int64_t)
                                  ((uint64_t)I_gap_open_offset << 32)
                                  | GAP_OPEN;

        const wfa_offset_t I_gap_extend_offset = prev_I_wf_e->offsets[k - 1] + 1;
        const wfa_backtrace_t I_gap_extend_bt = prev_I_wf_e->backtraces[k - 1];
        const int64_t I_gap_extend_offset_pb = (int64_t)
                                ((uint64_t)I_gap_extend_offset << 32)
                                | GAP_EXTEND;

        int64_t I_offset_pb = MAX_PB(I_gap_open_offset_pb,
                                     I_gap_extend_offset_pb);

        const wfa_offset_t I_offset = (wfa_offset_t)(I_offset_pb >> 32);
        I_wavefronts[curr_wf].offsets[k] = I_offset;

        // ~I backtraces
        // Include backtrace and previous backtrace offset
        const gap_op_t I_op = (gap_op_t)(I_offset_pb & 0xffffffff);
        wfa_backtrace_t I_backtrace;

        if (I_op == GAP_OPEN) {
            I_backtrace = I_gap_open_bt;
        } else {
            I_backtrace = I_gap_extend_bt;
        }

        I_backtrace.backtrace = (I_backtrace.backtrace << 2) | OP_INS;

        // TODO: Needed to offload ~I and ~D backtraces?
        // Offload ~I backtraces if the bitvector is full
        if (BT_IS_FULL(I_backtrace.backtrace)) {
            uint32_t prev = offload_backtrace(last_free_bt_position,
                                              I_backtrace,
                                              offloaded_backtraces);
            I_backtrace = {.backtrace = 0, .prev = prev};
        }

        I_wavefronts[curr_wf].backtraces[k] = I_backtrace;
        I_offset_pb = (uint64_t)(((uint64_t)I_offset << 32) | OP_INS);

        // ~D offsets
        const wfa_offset_t D_gap_open_offset = prev_wf_o->offsets[k + 1];
        const wfa_backtrace_t D_gap_open_bt = prev_wf_o->backtraces[k + 1];
        const int64_t D_gap_open_offset_pb = (int64_t)
                                  ((uint64_t)D_gap_open_offset << 32)
                                  | GAP_OPEN;

        const wfa_offset_t D_gap_extend_offset = prev_D_wf_e->offsets[k + 1];
        const wfa_backtrace_t D_gap_extend_bt = prev_D_wf_e->backtraces[k + 1];
        const int64_t D_gap_extend_offset_pb = (int64_t)
                                    ((uint64_t)D_gap_extend_offset << 32)
                                    | GAP_EXTEND;

        int64_t D_offset_pb = MAX_PB(D_gap_open_offset_pb,
                                     D_gap_extend_offset_pb);

        const wfa_offset_t D_offset = (wfa_offset_t)(D_offset_pb >> 32);
        D_wavefronts[curr_wf].offsets[k] = D_offset;

        // ~D backtraces
        const gap_op_t D_op = (gap_op_t)(D_offset_pb & 0xffffffff);
        wfa_backtrace_t D_backtrace;

        if (D_op == GAP_OPEN) {
            D_backtrace = D_gap_open_bt;
        } else {
            D_backtrace = D_gap_extend_bt;
        }

        D_backtrace.backtrace = (D_backtrace.backtrace << 2) | OP_DEL;

        // Offload ~D backtraces if the bitvector is full
        if (BT_IS_FULL(D_backtrace.backtrace)) {
            uint32_t prev = offload_backtrace(last_free_bt_position,
                                              D_backtrace,
                                              offloaded_backtraces);
            D_backtrace = {.backtrace = 0, .prev = prev};
        }

        D_wavefronts[curr_wf].backtraces[k] = D_backtrace;

        D_offset_pb = (uint64_t)(((uint64_t)D_offset << 32) | OP_DEL);

        // ~M update
        const wfa_offset_t X_offset = prev_wf_x->offsets[k] + 1;
        const wfa_backtrace_t X_backtrace = prev_wf_x->backtraces[k];
        const int64_t X_offset_pb = (int64_t)
                                     (((uint64_t)X_offset << 32)
                                     | OP_SUB);

        const int64_t M_offset_pb = MAX_PB(
                                        MAX_PB(X_offset_pb, D_offset_pb),
                                        I_offset_pb
                                        );
        // Extend
        wfa_offset_t M_offset = (wfa_offset_t)(M_offset_pb >> 32);
        M_offset = WF_extend_kernel(text, pattern, tlen, plen, k, M_offset);

        M_wavefronts[curr_wf].offsets[k] = M_offset;

        affine_op_t M_op = (affine_op_t)(M_offset_pb & 0xffffffff);
        wfa_backtrace_t M_backtrace;
        if (M_op == OP_INS) {
            M_backtrace = I_backtrace;
        } else if (M_op == OP_SUB) {
            M_backtrace = X_backtrace;
        } else {
            M_backtrace = D_backtrace;
        }

        M_backtrace.backtrace = (M_backtrace.backtrace << 2) | OP_SUB;

        // Offload backtraces if the bitvector is full
        if (BT_IS_FULL(M_backtrace.backtrace)) {
            //printf("OFFLOADING BACKTRACES!!!!!\n");
            uint32_t prev = offload_backtrace(last_free_bt_position,
                                              M_backtrace,
                                              offloaded_backtraces);
            M_backtrace = {.backtrace = 0, .prev = prev};
        }

        M_wavefronts[curr_wf].backtraces[k] = M_backtrace;
    }

    if (threadIdx.x == 0) {
        M_wavefronts[curr_wf].hi = hi;
        M_wavefronts[curr_wf].lo = lo;
        M_wavefronts[curr_wf].exist = true;

        I_wavefronts[curr_wf].hi = hi;
        I_wavefronts[curr_wf].lo = lo;
        I_wavefronts[curr_wf].exist = true;

        D_wavefronts[curr_wf].hi = hi;
        D_wavefronts[curr_wf].lo = lo;
        D_wavefronts[curr_wf].exist = true;
    }
}

__device__ void update_curr_wf (wfa_wavefront_t* M_wavefronts,
                                wfa_wavefront_t* I_wavefronts,
                                wfa_wavefront_t* D_wavefronts,
                                const int active_working_set_size,
                                const int max_wf_size,
                                int* curr_wf) {
    // As we read wavefronts "forward" in the waveronts arrays, so the wavefront
    // index is moved backwards.
    const int wf_idx = (*curr_wf - 1 + active_working_set_size) % active_working_set_size;

    // TODO: Check if this is necessary in some penalties combination
    // Set new wf to NULL, as new wavefront may be smaller than the
    // previous one
    //wfa_offset_t* to_clean_M = M_wavefronts[wf_idx].offsets - (max_wf_size/2);
    M_wavefronts[wf_idx].exist = false;

    //wfa_offset_t* to_clean_I = I_wavefronts[wf_idx].offsets - (max_wf_size/2);
    I_wavefronts[wf_idx].exist = false;

    //wfa_offset_t* to_clean_D = D_wavefronts[wf_idx].offsets - (max_wf_size/2);
    D_wavefronts[wf_idx].exist = false;

    //for (int i=threadIdx.x; i<max_wf_size; i+=blockDim.x) {
    //    to_clean_M[i] = -1;
    //    to_clean_D[i] = -1;
    //    to_clean_I[i] = -1;
    //}

    *curr_wf = wf_idx;

}

__global__ void alignment_kernel (
                            const char* packed_sequences_buffer,
                            const sequence_pair_t* sequences_metadata,
                            const size_t num_alignments,
                            const int max_steps,
                            uint8_t* const wf_data_buffer,
                            const affine_penalties_t penalties,
                            wfa_backtrace_t* offloaded_backtraces_global,
                            wfa_backtrace_t* offloaded_backtraces_results,
                            alignment_result_t* results) {
    const int tid = threadIdx.x;
    // m = 0 for WFA
    const int x = penalties.x;
    const int o = penalties.o;
    const int e = penalties.e;

    const sequence_pair_t curr_batch_alignment_base = sequences_metadata[0];
    const size_t base_offset_packed = curr_batch_alignment_base.pattern_offset_packed;

    const sequence_pair_t metadata = sequences_metadata[blockIdx.x];
    const char* text_global = packed_sequences_buffer + metadata.text_offset_packed - base_offset_packed;
    const char* pattern_global = packed_sequences_buffer + metadata.pattern_offset_packed - base_offset_packed ;
    const int tlen = metadata.text_len;
    const int plen = metadata.pattern_len;

    // 2KiB to test
    __shared__ char sequences_sh[2048];
    //for (int i=threadIdx.x; i<2048; i+=blockDim.x) {
    //    sequences_sh[i] = 0;
    //}
    //__syncthreads();

    // Sequence buffers are 32 bits aligned
    const int p_buf_packed_len = (plen + (4 - (plen % 4))) / 4;
    const int t_buf_packed_len = (tlen + (4 - (tlen % 4))) / 4;

    char* pattern = &sequences_sh[0];
    char* text = &sequences_sh[1024];

    for (int i=threadIdx.x*4; i<p_buf_packed_len+4; i+=blockDim.x*4) {
        *(uint32_t*)(&pattern[i]) = *(uint32_t*)(&pattern_global[i]);
    }
    for (int i=threadIdx.x*4; i<t_buf_packed_len+4; i+=blockDim.x*4) {
        *(uint32_t*)(&text[i]) = *(uint32_t*)(&text_global[i]);
    }
    //__syncthreads();

    // TODO: Move to function/macro + use in lib/sequence_alignment.cu
    size_t bt_offloaded_size = BT_OFFLOADED_ELEMENTS(max_steps);
    wfa_backtrace_t* const offloaded_backtraces = \
             &offloaded_backtraces_global[blockIdx.x * bt_offloaded_size];

    size_t bt_results_size = BT_OFFLOADED_RESULT_ELEMENTS(max_steps);
    wfa_backtrace_t* const offloaded_backtrace_results_base = \
             &offloaded_backtraces_results[blockIdx.x * bt_results_size];

    // In shared memory:
    // - Wavefronts needed to calculate current WF_s, there are 3 "pyramids" so
    //   this number of wavefront is 3 times (WF_{max(o+e, x)} --> WF_s)
    extern __shared__ char sh_mem[];

    // TODO: +1 because of the current wf?
    const int active_working_set_size = MAX(o+e, x) + 1;
    const int max_wf_size = 2 * max_steps + 1;

    // Offsets and backtraces must be 32 bits aligned to avoid unaligned access
    // errors on the structs
    int offsets_size = active_working_set_size * max_wf_size;
    offsets_size = offsets_size + (4 - (offsets_size % 4));

    int bt_size = active_working_set_size * max_wf_size;
    bt_size = bt_size + (4 - (bt_size % 4));

    const size_t wf_data_buffer_size =
                    // Offsets space
                    (offsets_size * 3 * sizeof(wfa_offset_t))
                    // Backtraces space
                    + (bt_size * 3 * sizeof(wfa_backtrace_t));
    uint8_t* curr_alignment_wf_data_buffer = wf_data_buffer
                                             + (wf_data_buffer_size * blockIdx.x);

    wfa_offset_t* M_base = (wfa_offset_t*)curr_alignment_wf_data_buffer;
    wfa_offset_t* I_base = M_base + offsets_size;
    wfa_offset_t* D_base = I_base + offsets_size;

    wfa_backtrace_t* M_bt_base = (wfa_backtrace_t*)(D_base + offsets_size);
    wfa_backtrace_t* I_bt_base = M_bt_base + bt_size;
    wfa_backtrace_t* D_bt_base = I_bt_base + bt_size;

    // Wavefronts structres reside in shared
    wfa_wavefront_t* M_wavefronts = (wfa_wavefront_t*)sh_mem;
    wfa_wavefront_t* I_wavefronts = (M_wavefronts + active_working_set_size);
    wfa_wavefront_t* D_wavefronts = (I_wavefronts + active_working_set_size);

    uint32_t* last_free_bt_position = (uint32_t*)
                                          (D_wavefronts + active_working_set_size);

    // Start at 1 because 0 is used as NULL (no more backtraces blocks to
    // recover)
    *last_free_bt_position = 1;

    // Initialize all wavefronts to -1
    for (int i=tid; i<(offsets_size * 3); i+=blockDim.x) {
        M_base[i] = -1;
    }

    for (int i=tid; i<(bt_size * 3); i+=blockDim.x) {
        M_bt_base[i] = {0};
    }

    // Initialize wavefronts memory
    for (int i=tid; i<active_working_set_size; i+=blockDim.x) {
        M_wavefronts[i].offsets = M_base + (i * max_wf_size) + (max_wf_size/2);
        M_wavefronts[i].backtraces = M_bt_base + (i * max_wf_size) + (max_wf_size/2);
        M_wavefronts[i].hi = 0;
        M_wavefronts[i].lo = 0;
        M_wavefronts[i].exist = false;

        I_wavefronts[i].offsets = I_base + (i * max_wf_size) + (max_wf_size/2);
        I_wavefronts[i].backtraces = I_bt_base + (i * max_wf_size) + (max_wf_size/2);
        I_wavefronts[i].hi = 0;
        I_wavefronts[i].lo = 0;
        I_wavefronts[i].exist = false;

        D_wavefronts[i].offsets = D_base + (i * max_wf_size) + (max_wf_size/2);
        D_wavefronts[i].backtraces = D_bt_base + (i * max_wf_size) + (max_wf_size/2);
        D_wavefronts[i].hi = 0;
        D_wavefronts[i].lo = 0;
        D_wavefronts[i].exist = false;
    }

    __syncthreads();

    int curr_wf = 0;

    if (tid == 0) {
        wfa_offset_t initial_ext = WF_extend_kernel(
            text,
            pattern,
            tlen, plen,
            0, 0);
        M_wavefronts[curr_wf].offsets[0] = initial_ext;
        M_wavefronts[curr_wf].exist = true;
    }

    __syncthreads();

    // TODO: Change tarket K if we don't start form WF 0 (cooperative strategy)
    const int target_k = EWAVEFRONT_DIAGONAL(tlen, plen);
    const int target_k_abs = (target_k >= 0) ? target_k : -target_k;
    const wfa_offset_t target_offset = EWAVEFRONT_OFFSET(tlen, plen);

    bool finished = false;

    int distance = 0;
    // steps = number of editions
    int steps = 0;
    // TODO: target_k_abs <= distance or <= steps (?)
    if (!(target_k_abs <= distance && M_wavefronts[curr_wf].exist && M_wavefronts[curr_wf].offsets[target_k] == target_offset)) {

        update_curr_wf(
            M_wavefronts,
            I_wavefronts,
            D_wavefronts,
            active_working_set_size,
            max_wf_size,
            &curr_wf);

        distance++;
        steps++;
        __syncthreads();

        while (steps < (max_steps - 1)) {

            bool M_exist = false;
            bool GAP_exist = false;
            const int o_delta = (curr_wf + o + e) % active_working_set_size;
            const int e_delta = (curr_wf + e) % active_working_set_size;
            const int x_delta = (curr_wf + x) % active_working_set_size;
            if ((distance - o - e) >= 0) {
                // Just test with I because I and D exist in the same distances
                GAP_exist = M_wavefronts[o_delta].exist
                          || I_wavefronts[e_delta].exist;
            }

            if (GAP_exist) {
                M_exist = true;
            } else {
                if ((distance - x) >= 0) {
                    M_exist = M_wavefronts[x_delta].exist;
                } 
            }

            if (!GAP_exist && !M_exist) {
                distance++;
            } else {
                if (M_exist && !GAP_exist) {
                    next_M(M_wavefronts, curr_wf, active_working_set_size, x,
                           text, pattern, tlen, plen,
                           last_free_bt_position, offloaded_backtraces);
                } else {
                    next_MDI(
                        M_wavefronts, I_wavefronts, D_wavefronts,
                        curr_wf, active_working_set_size,
                        x, o, e,
                        text, pattern, tlen, plen,
                        last_free_bt_position, offloaded_backtraces);

                    // Wavefront only grows if there's an operation in the ~I or
                    // ~D matrices
                    steps++;
                }

                // TODO: This is necessary for now, try to find a less sync
                // version
                __syncthreads();

                if (target_k_abs <= distance && M_exist && M_wavefronts[curr_wf].offsets[target_k] == target_offset) {
                    finished = true;
                    break;
                }

                distance++;
            }

        update_curr_wf(
            M_wavefronts,
            I_wavefronts,
            D_wavefronts,
            active_working_set_size,
            max_wf_size,
            &curr_wf);

        __syncthreads();
        }
    } else {
        finished = true;
    }

    if  (tid == 0) {
        results[blockIdx.x].distance = distance;
        results[blockIdx.x].finished = finished;
        results[blockIdx.x].backtrace = M_wavefronts[curr_wf].backtraces[target_k];

        wfa_backtrace_t* curr_result = &M_wavefronts[curr_wf].backtraces[target_k];

        // Save the list in reversed order
        int i = 0;
        while (curr_result->prev != 0) {
            offloaded_backtrace_results_base[i] = \
                                        offloaded_backtraces[curr_result->prev];
            curr_result = &offloaded_backtraces[curr_result->prev];
            i++;
        }

        results[blockIdx.x].num_bt_blocks = i;
    }
}
